#include "hip/hip_runtime.h"
#include "./c_runtime_api.h"
#include <cassert>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

__global__ void array_set_kernel(int nrow, int ncol, 
		                 float* input, 
				 float value) {
  int c_ = blockIdx.x * blockDim.x + threadIdx.x;
  int r_ = blockIdx.y * blockDim.y + threadIdx.y;
  if (r_ >= nrow || c_ >= ncol)  
    return;
  input[r_ * ncol + c_] = value;
}

int DLGpuArraySet(DLArrayHandle arr, float value) {
  int nrow = arr->shape[0];
  int ncol = arr->shape[1];

  float* input_data = (float*)arr->data;

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((ncol + dimBlock.x - 1) / dimBlock.x,
	       (nrow + dimBlock.y - 1) / dimBlock.y);
  array_set_kernel<<<dimGrid, dimBlock>>>(nrow, ncol, input_data, value);
  return 0;
}


__global__ void broadcast_to_kernel(int ntimes, int nnum,
				    const float* input,
				    float* output) {
  int n = blockDim.x * blockIdx.x + threadIdx.x;
  if (n >= nnum)
    return;
  for (int i = 0; i < ntimes; i++) {
    float* output_ = output + nnum * i;
    output_[n] = input[n];
  } 
}

int DLGpuBroadcastTo(const DLArrayHandle input, DLArrayHandle output) {
  int ntimes = output->shape[0];
  int nnum = 1;
  for (int i = 1; i < output->ndim; i++)
    nnum *= output->shape[i];
  
  const float* input_data = (const float*)input->data;
  float* output_data = (float*)output->data;

  dim3 dimBlock(BLOCK_SIZE);
  dim3 dimGrid((nnum + dimBlock.x - 1) / dimBlock.x);
  
  broadcast_to_kernel<<<dimGrid, dimBlock>>>(
      ntimes, nnum, input_data, output_data);
  return 0;
}


__global__ void reduce_sum_axis_zero_kernel(int reduce_n, int remain_n,
		                            const float* input,
					    float* output) {
  int n = blockDim.x * blockIdx.x + threadIdx.x;
  if (n >= remain_n)
    return;
  
  float sum = 0.0;
  for (int i = 0; i < reduce_n; i++) {
    const float* input_ = input + remain_n * i;
    sum += input_[n];
  }
  output[n] = sum;
}

int DLGpuReduceSumAxisZero(const DLArrayHandle input, DLArrayHandle output) {
  int reduce_n = input->shape[0];
  int remain_n = 1;
  for (int i = 1; i < input->ndim; i++) {
    remain_n *= input->shape[i];
  }
  const float* input_data = (const float*)input->data;
  float* output_data = (float*)output->data;

  dim3 dimBlock(BLOCK_SIZE);
  dim3 dimGrid((remain_n + dimBlock.x- 1) / dimBlock.x);
  reduce_sum_axis_zero_kernel<<<dimGrid, dimBlock>>>(
      reduce_n, remain_n, input_data, output_data);
  return 0;
}


__global__ void matrix_elementwise_add_kernel(int nrow, int ncol, 
					      const float* input_a, 
					      const float* input_b, 
					      float* output) {
  int r_ = blockDim.y * blockIdx.y + threadIdx.y;
  int c_ = blockDim.x * blockIdx.x + threadIdx.x;
  if (r_ >= nrow || c_ >= ncol)
    return;
  output[r_ * ncol + c_] = input_a[r_ * ncol + c_] + input_b[r_ * ncol + c_];
}

int DLGpuMatrixElementwiseAdd(const DLArrayHandle matA,
                              const DLArrayHandle matB, 
			      DLArrayHandle output) {
  int nrow = matA->shape[0];
  int ncol = matA->shape[1];
 
  const float* input_data_a = (const float*)matA->data;
  const float* input_data_b = (const float*)matB->data;
  float* output_data = (float*)output->data; 

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((ncol + dimBlock.x - 1) / dimBlock.x, 
	       (nrow + dimBlock.y - 1) / dimBlock.y);
  
  matrix_elementwise_add_kernel<<<dimGrid, dimBlock>>>(
      nrow, ncol, input_data_a, input_data_b, output_data); 
  return 0;
}


__global__ void matrix_elementwise_add_const_kernel(int nrow, int ncol, 
						    const float* input, 
						    float val, 
						    float* output) {
  int r_ = blockDim.y * blockIdx.y + threadIdx.y;
  int c_ = blockDim.x * blockIdx.x + threadIdx.x;
  if (r_ >= nrow || c_ >= ncol)
    return;
  output[r_ * ncol + c_] = input[r_ * ncol + c_] + val; 
}

int DLGpuMatrixElementwiseAddByConst(const DLArrayHandle input, 
				     float val,
                                     DLArrayHandle output) {
  int nrow = input->shape[0];
  int ncol = input->shape[1];
  
  const float* input_data = (const float*)input->data;
  float* output_data = (float*)output->data; 
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((ncol + dimBlock.x - 1) / dimBlock.x, 
	       (nrow + dimBlock.y - 1) / dimBlock.y);
  
  matrix_elementwise_add_const_kernel<<<dimGrid, dimBlock>>>(
    nrow, ncol, input_data, val, output_data);
  return 0;
}


__global__ void matrix_elementwise_multiply_kernel(int nrow, int ncol, 
						   const float* input_a, 
						   const float* input_b, 
						   float* output) {
  int r_ = blockDim.y * blockIdx.y + threadIdx.y;
  int c_ = blockDim.x * blockIdx.x + threadIdx.x;
  if (r_ >= nrow || c_ >= ncol)
    return;
  output[r_ * ncol + c_] = input_a[r_ * ncol + c_] * input_b[r_ * ncol + c_];
}

int DLGpuMatrixElementwiseMultiply(const DLArrayHandle matA,
                                   const DLArrayHandle matB,
                                   DLArrayHandle output) {
  int nrow = matA->shape[0];
  int ncol = matA->shape[1];
 
  const float* input_data_a = (const float*)matA->data;
  const float* input_data_b = (const float*)matB->data;
  float* output_data = (float*)output->data;

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((ncol + dimBlock.x - 1) / dimBlock.x, 
	       (nrow + dimBlock.y - 1) / dimBlock.y);
  
  matrix_elementwise_multiply_kernel<<<dimBlock, dimGrid>>>(
      nrow, ncol, input_data_a, input_data_b, output_data); 
  return 0;
}


__global__ void matrix_multiply_const(int nrow, int ncol, 
				      const float* input, 
				      float val, 
				      float* output) {
  int r_ = blockDim.y * blockIdx.y + threadIdx.y;
  int c_ = blockDim.x * blockIdx.x + threadIdx.x;
  if (r_ >= nrow || c_ >= ncol)
    return;
  output[r_ * ncol + c_] = input[r_ * ncol + c_] * val; 

}

int DLGpuMatrixMultiplyByConst(const DLArrayHandle input, 
			       float val,
                               DLArrayHandle output) {
  int nrow = input->shape[0];
  int ncol = input->shape[1];
  
  const float* input_data = (const float*)input->data;
  float* output_data = (float*)output->data;

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((ncol + dimBlock.x - 1) / dimBlock.x, 
	       (nrow + dimBlock.y - 1) / dimBlock.y);

  matrix_multiply_const<<<dimGrid, dimBlock>>>(
      nrow, ncol, input_data, val, output_data);
  return 0;
}

__global__ void matrix_multiply_kernel(const float* input_a, 
				       const float* input_b,
				       bool transposeA,
				       bool transposeB,
				       int nrow_a, int ncol_a,
				       int nrow_b, int ncol_b, 
				       int nrow, int ncol, int nwidth,
				       float* output) {
  int r_ = blockDim.y * blockIdx.y + threadIdx.y;
  int c_ = blockDim.x * blockIdx.x + threadIdx.x;
  if (r_ >= nrow || c_ >= ncol)
    return;

  float a = 0.0, b = 0.0, sum = 0.0;
  for (int i = 0; i < nwidth; i++) {
      a = transposeA ? input_a[ncol_a * i + r_] : input_a[ncol_a * r_ + i];
      b = transposeB ? input_b[ncol_b * c_ + i] : input_b[ncol_b * i + c_];
      sum += a * b;
  }
  output[ncol * r_ + c_] = sum;
  return;
}

int DLGpuMatrixMultiply(const DLArrayHandle matA, bool transposeA,
                        const DLArrayHandle matB, bool transposeB,
                        DLArrayHandle matC) {
  int nrow_a = matA->shape[0], ncol_a = matA->shape[1];
  int nrow_b = matB->shape[0], ncol_b = matB->shape[1];

  // nrow is the number of row of result matrix;
  int nrow = transposeA ? ncol_a : nrow_a;
  // ncol is the number of col of result matrix;
  int ncol = transposeB ? nrow_b : ncol_b;
  int nwidth = transposeA ? nrow_a : ncol_a;

  const float* input_data_a = (const float*)matA->data;
  const float* input_data_b = (const float*)matB->data;
  float* output_data = (float*)matC->data;
  
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((ncol + dimBlock.x - 1) / dimBlock.x,
	       (nrow + dimBlock.y - 1) / dimBlock.y);

  matrix_multiply_kernel<<<dimGrid, dimBlock>>>(
      input_data_a, input_data_b, 
      transposeA, transposeB,
      nrow_a, ncol_a, nrow_b, ncol_b, 
      nrow, ncol, nwidth,
      output_data);
  return 0;
}

__global__ void relu_kernel(int nrow, int ncol, 
			    const float* input, 
			    float* output) {
  int r_ = blockDim.y * blockIdx.y + threadIdx.y;
  int c_ = blockDim.x * blockIdx.x + threadIdx.x;

  if (r_ >= nrow || c_ >= ncol)
    return;
  float val = input[r_ * ncol + c_];
  output[r_ * ncol + c_] = val >= 0 ? val : 0;
}

int DLGpuRelu(const DLArrayHandle input, DLArrayHandle output) {
  int nrow = input->shape[0];
  int ncol = input->shape[1];

  const float* input_data = (const float*)input->data;
  float* output_data = (float*)output->data;

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((ncol+ dimBlock.x - 1) / dimBlock.x, 
	       (nrow + dimBlock.y - 1) / dimBlock.y);
  relu_kernel<<<dimGrid, dimBlock>>>(
      nrow, ncol, input_data, output_data);
  return 0;
}

__global__ void relu_gradient_kernel(int nrow, int ncol, 
				     const float* input, 
				     const float* in_grad, 
				     float* output) {
  int r_ = blockDim.y * blockIdx.y + threadIdx.y;
  int c_ = blockDim.x * blockIdx.x + threadIdx.x;

  if (r_ >= nrow || c_ >= ncol)
    return;

  float input_val = input[r_ * ncol + c_];
  float in_grad_val = in_grad[r_ * ncol + c_];
  
  output[r_ * ncol + c_] = input_val >= 0 ? in_grad_val : 0;
}

int DLGpuReluGradient(const DLArrayHandle input, 
		      const DLArrayHandle in_grad,
                      DLArrayHandle output) {
  int nrow = input->shape[0];
  int ncol = input->shape[1];

  const float* input_data =  (const float*)input->data;
  const float* in_grad_data =  (const float*)in_grad->data;
  float* output_data = (float*)output->data;

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((ncol + dimBlock.x - 1) / dimBlock.x, 
	       (nrow + dimBlock.y - 1) / dimBlock.y);
  relu_gradient_kernel<<<dimGrid, dimBlock>>>(
      nrow, ncol, 
      input_data,
      in_grad_data,
      output_data);
  return 0;
}

__global__ void softmax_kernel(int nrow, int ncol,
		               const float* input,
			       float* output) {
  int r_ = blockDim.x * blockIdx.x + threadIdx.x;
  if (r_ >= nrow)
    return;

  input += r_ * ncol;
  float min_val = input[0];
  for (int i = 1; i < ncol; i++)
    min_val = min(min_val, input[i]);

  float sum = 0.0;
  for (int i = 0; i < ncol; i++) 
    sum += exp(input[i] - min_val);

  output += r_ * ncol;
  for (int i = 0; i < ncol; i++)
    output[i] = exp(input[i] - min_val) / sum;
}

int DLGpuSoftmax(const DLArrayHandle input, DLArrayHandle output) {
  int nrow = input->shape[0];
  int ncol = input->shape[1];
  const float* input_data = (const float*)input->data; 
  float* output_data = (float*)output->data;
  dim3 dimBlock(BLOCK_SIZE);
  dim3 dimGrid((nrow + dimBlock.x - 1) / dimBlock.x);

  softmax_kernel<<<dimGrid, dimBlock>>>(
      nrow, ncol, input_data, output_data); 
  return 0;
}

// y = inputs[0], y_ = inputs[1]
// np.mean(-np.sum(y_ * np.log(softmax(y)), axis=1), keepdims=True)
__global__ void matrix_softmax_cross_entropy_kernel(int nrow, int ncol,
                                                    const float *input_a,
                                                    const float *input_b,
                                                    float *output) {
  // Dynamic shared memory, size provided at kernel launch.
  extern __shared__ float loss_per_row[];
  // Two dimensional thread blocks.
  int y = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
          threadIdx.x;
  if (y >= nrow) {
    return;
  }
  input_a += y * ncol;
  input_b += y * ncol;
  float maxval = *input_a;
  // Find max for a row.
  for (int x = 1; x < ncol; ++x) {
    maxval = max(maxval, input_a[x]);
  }
  // Deduct by max for a row, and raise to exp.
  float sum = 0;
  for (int x = 0; x < ncol; ++x) {
    sum += exp(input_a[x] - maxval);
  }
  // Compute per-row loss.
  float loss = 0;
  for (int x = 0; x < ncol; ++x) {
    loss -= input_b[x] * log(exp(input_a[x] - maxval) / sum);
  }
  loss_per_row[y] = loss;
  __syncthreads();
  // Compute reduce_mean across rows.
  float mean_loss = 0;
  // Use a single thread to reduce mean across rows.
  if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
    for (int i = 0; i < nrow; ++i) {
      mean_loss += loss_per_row[i];
    }
    mean_loss /= nrow;
    output[0] = mean_loss;
  }
}

int DLGpuSoftmaxCrossEntropy(const DLArrayHandle input_a,
                             const DLArrayHandle input_b,
                             DLArrayHandle output) {
  assert(input_a->ndim == 2);
  assert(input_b->ndim == 2);
  assert(output->ndim == 1);
  assert(input_a->shape[0] == input_b->shape[0] &&
         input_a->shape[1] == input_b->shape[1]);
  int nrow = input_a->shape[0];
  // Maximum x- or y-dimension of a block = 1024
  // But we need 'nrow' shared memory, and max shared memory is 48KB.
  // Conservatively allow max 16KB shared memory.
  assert(nrow <= 1024 * 4);
  int ncol = input_a->shape[1];
  const float *input_data_a = (const float *)input_a->data;
  const float *input_data_b = (const float *)input_b->data;
  float *output_data = (float *)output->data;
  dim3 threads;
  if (nrow <= 1024) {
    threads.x = nrow;
  } else {
    threads.x = 1024;
    threads.y = (nrow + 1023) / 1024;
  }
  // 1 block, each block with 'threads' number of threads with 'nrow' shared
  // memory size
  matrix_softmax_cross_entropy_kernel<<<1, threads, nrow * sizeof(float)>>>(
      nrow, ncol, input_data_a, input_data_b, output_data);
  return 0;
}
